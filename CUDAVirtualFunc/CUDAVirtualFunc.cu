#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "helper.h"

template <typename T>
__global__ void testKernel(T val)
{
    printf("\n my value = %d\n", val);
}

template <typename T>
struct Base
{
    Base(){
        hipStreamCreate(&m_stream);
        m_value = 77;
    };

    virtual void launch() = 0;

protected:
    hipStream_t m_stream;
    T m_value;
};

template<typename T>
struct Derived: public Base<T>
{
    Derived() : Base<T>(){

    }

    virtual void launch() override{
        testKernel<T><<<1,1,0, this->m_stream>>>(this->m_value);
    }
};



int main(int argc, char **argv)
{     
    Derived<int> d;
    d.launch();

    CUDA_ERROR(hipDeviceSynchronize());
    return EXIT_SUCCESS;
}

